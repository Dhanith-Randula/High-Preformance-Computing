#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define MAX_LEN 6
#define CHARSET "0123456789ABCDEFGHIJKLMNOPQRSTUVWXYZabcdefghijklmnopqrstuvwxyz"
#define CHARSET_LEN 62
#include <omp.h>
int found_length = 0;
char found_result[MAX_LEN + 1] = {0};
bool found = false;

// GPU constants
__device__ __constant__ char d_charset[CHARSET_LEN];
__device__ __constant__ char d_target_hash[65];
__device__ volatile unsigned int d_found = 0;
__device__ char d_result[MAX_LEN + 1];

// Converts an index into a password string
__device__ void index_to_password(unsigned long long idx, int length, char *out) {
    for (int i = length - 1; i >= 0; i--) {
        out[i] = d_charset[idx % CHARSET_LEN];
        idx /= CHARSET_LEN;
    }
    out[length] = '\0';
}

// Compare two hex hashes
__device__ bool match_hash(const char *h1, const char *h2) {
    for (int i = 0; i < 64; i++) {
        if (h1[i] != h2[i]) return false;
    }
    return true;
}

// SHA-256 constants
__device__ __constant__ unsigned int k[64] = {
    0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
    0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
    0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
    0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
    0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
    0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
    0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
    0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2
};

// SHA-256 helper functions
__device__ inline unsigned int rotr(unsigned int x, unsigned int n) {
    return (x >> n) | (x << (32 - n));
}
__device__ inline unsigned int Ch(unsigned int x, unsigned int y, unsigned int z) {
    return (x & y) ^ (~x & z);
}
__device__ inline unsigned int Maj(unsigned int x, unsigned int y, unsigned int z) {
    return (x & y) ^ (x & z) ^ (y & z);
}
__device__ inline unsigned int Sigma0(unsigned int x) {
    return rotr(x, 2) ^ rotr(x, 13) ^ rotr(x, 22);
}
__device__ inline unsigned int Sigma1(unsigned int x) {
    return rotr(x, 6) ^ rotr(x, 11) ^ rotr(x, 25);
}
__device__ inline unsigned int sigma0(unsigned int x) {
    return rotr(x, 7) ^ rotr(x, 18) ^ (x >> 3);
}
__device__ inline unsigned int sigma1(unsigned int x) {
    return rotr(x, 17) ^ rotr(x, 19) ^ (x >> 10);
}

// SHA-256 padding and message schedule
__device__ void sha256_pad(const char *input, int len, unsigned int *w) {
    for (int i = 0; i < 16; i++) w[i] = 0;
    for (int i = 0; i < len; i++) {
        w[i >> 2] |= ((unsigned int)input[i]) << (24 - (8 * (i & 3)));
    }
    w[len >> 2] |= 0x80 << (24 - (8 * (len & 3)));
    unsigned long long bit_len = (unsigned long long)len * 8;
    w[15] = (unsigned int)(bit_len);
    w[14] = (unsigned int)(bit_len >> 32);
}

// Computes SHA-256 hash of a string
__device__ void sha256(const char *input, int len, char *output) {
    unsigned int h[8] = {
        0x6a09e667, 0xbb67ae85, 0x3c6ef372, 0xa54ff53a,
        0x510e527f, 0x9b05688c, 0x1f83d9ab, 0x5be0cd19
    };

    unsigned int w[64];
    sha256_pad(input, len, w);

    for (int i = 16; i < 64; i++) {
        w[i] = sigma1(w[i - 2]) + w[i - 7] + sigma0(w[i - 15]) + w[i - 16];
    }

    unsigned int a = h[0], b = h[1], c = h[2], d = h[3];
    unsigned int e = h[4], f = h[5], g = h[6], h_var = h[7];

    for (int i = 0; i < 64; i++) {
        unsigned int T1 = h_var + Sigma1(e) + Ch(e, f, g) + k[i] + w[i];
        unsigned int T2 = Sigma0(a) + Maj(a, b, c);
        h_var = g;
        g = f;
        f = e;
        e = d + T1;
        d = c;
        c = b;
        b = a;
        a = T1 + T2;
    }

    h[0] += a; h[1] += b; h[2] += c; h[3] += d;
    h[4] += e; h[5] += f; h[6] += g; h[7] += h_var;

    for (int i = 0; i < 8; i++) {
        for (int j = 0; j < 4; j++) {
            unsigned char byte = (h[i] >> (24 - j * 8)) & 0xFF;
            output[i * 8 + j * 2] = (byte >> 4) < 10 ? '0' + (byte >> 4) : 'a' + (byte >> 4) - 10;
            output[i * 8 + j * 2 + 1] = (byte & 0xF) < 10 ? '0' + (byte & 0xF) : 'a' + (byte & 0xF) - 10;
        }
    }
    output[64] = '\0';
}

// Kernel to crack passwords
__global__ void crack_password(int length, unsigned long long start_idx, unsigned long long end_idx) {
    unsigned long long idx = start_idx + blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= end_idx || d_found) return;

    char candidate[MAX_LEN + 1];
    char hash_out[65];

    index_to_password(idx, length, candidate);
    sha256(candidate, length, hash_out);

    if (match_hash(hash_out, d_target_hash)) {
        unsigned int expected = 0;
        if (atomicCAS((unsigned int*)&d_found, expected, 1) == 0) {
            for (int i = 0; i < length; i++) {
                d_result[i] = candidate[i];
            }
            d_result[length] = '\0';
        }
    }
}

int main() {
    const char target_hash[65] = "24f916304a3a9fc2213185a5bce0c723813a6425bb26958647059b208f3f5420";
    
    // Copy constant data to GPU once
    hipMemcpyToSymbol(HIP_SYMBOL(d_charset), CHARSET, CHARSET_LEN);
    hipMemcpyToSymbol(HIP_SYMBOL(d_target_hash), target_hash, 65);

    printf("Target hash: %s\n", target_hash);
    printf("Searching passwords length 1 to %d...\n", MAX_LEN);

    double total_start = omp_get_wtime();  // Total time start

    #pragma omp parallel for schedule(dynamic) // according to load divide to threads
    for (int length = 1; length <= MAX_LEN; length++) {
        if (found) continue;  // Early exit if already found
        // Calculate combinations for this length
        unsigned long long combinations = 1;
        for (int i = 0; i < length; i++) combinations *= CHARSET_LEN;
        
        printf("\nTrying length %d: %llu combinations\n", length, combinations);

        // Reset found flag
        unsigned int h_found = 0;
        hipMemcpyToSymbol(HIP_SYMBOL(d_found), &h_found, sizeof(unsigned int));

        // Setup timing
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);

        // Configure kernel launch
        int threads_per_block = 256;
        unsigned long long max_blocks = 65535;
        unsigned long long chunk_size = threads_per_block * max_blocks;
        unsigned long long chunks = (combinations + chunk_size - 1) / chunk_size;

        for (unsigned long long chunk = 0; chunk < chunks && !h_found; chunk++) {
            unsigned long long start_idx = chunk * chunk_size;
            unsigned long long end_idx = (chunk + 1) * chunk_size;
            if (end_idx > combinations) end_idx = combinations;
            
            int blocks = (end_idx - start_idx + threads_per_block - 1) / threads_per_block;
            crack_password<<<blocks, threads_per_block>>>(length, start_idx, end_idx);
            
            hipDeviceSynchronize();  // Ensure GPU work is done before checking flag
            hipMemcpyFromSymbol(&h_found, HIP_SYMBOL(d_found), sizeof(unsigned int));
        }

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);

        printf("Length %d: Time taken = %.3f seconds\n", length, milliseconds / 1000.0f);

       
        

        if (h_found) {
        #pragma omp critical
        {
            if (!found) {
                hipMemcpyFromSymbol(found_result, HIP_SYMBOL(d_result), length + 1);
                found = true;
                found_length = length;
            }
        }
    }
    }

    if (found) {
            printf("\nPASSWORD FOUND: %s\n", found_result);
            double total_end = omp_get_wtime();
            printf("\nTotal Time Taken: %.3f seconds\n", total_end - total_start);

        }
    else {
        printf("\nPassword not found in lengths 1-%d\n", MAX_LEN);
        double total_end = omp_get_wtime();
        printf("\nTotal Time Taken: %.3f seconds\n", total_end - total_start);
    }
}


//nvcc CUDACrack.cu -o CUDACrack.o
// nvcc -arch=sm_75 -Xcompiler -fopenmp hybride.cu -o hybrid.o
